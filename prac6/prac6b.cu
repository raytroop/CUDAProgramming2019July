//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>


//
// template kernel routine
//

template  <typename T>
__global__ void my_first_kernel(T *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = threadIdx.x;
}


//
// CUDA routine to be called by main code
//

extern
int prac6(int nblocks, int nthreads)
{
  float *h_x, *d_x;
  int   *h_i, *d_i;
  double *h_xx, *d_xx;
  int   nsize, n;

  // allocate memory for arrays

  nsize = nblocks*nthreads ;

  h_xx = (double *)malloc(nsize*sizeof(double));
  hipMalloc((void **)&d_xx, nsize*sizeof(double));

  h_x = (float *)malloc(nsize*sizeof(float));
  hipMalloc((void **)&d_x, nsize*sizeof(float));

  h_i = (int *)malloc(nsize*sizeof(int));
  hipMalloc((void **)&d_i, nsize*sizeof(int));

  // execute kernel for double
  my_first_kernel<<<nblocks,nthreads>>>(d_xx);
  hipMemcpy(h_xx,d_xx,nsize*sizeof(double),hipMemcpyDeviceToHost);
  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_xx[n]);

  // execute kernel for float
  my_first_kernel<<<nblocks,nthreads>>>(d_x);
  hipMemcpy(h_x,d_x,nsize*sizeof(float),hipMemcpyDeviceToHost);
  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // execute kernel for ints
  my_first_kernel<<<nblocks,nthreads>>>(d_i);
  hipMemcpy(h_i,d_i,nsize*sizeof(int),hipMemcpyDeviceToHost);
  for (n=0; n<nsize; n++) printf(" n,  i  =  %d  %d \n",n,h_i[n]);

  // free memory
  hipFree(d_xx);
  free(h_xx);
  hipFree(d_x);
  free(h_x);
  hipFree(d_i);
  free(h_i);

  return 0;
}


